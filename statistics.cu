#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "statistics.h"
#include "statistics_kernel.cu"

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

Statistics::Statistics(int samples, int bootstraps)
{
    NSAMPLES = samples;
	NBOOTSTRAPS = bootstraps;
}

Statistics::~Statistics()
{

}

void Statistics::SetInputDataPointers(float* data)
{
	h_Data = data;
}

void Statistics::SetOutputDataPointers(float* means)
{
	h_Means = means;
}



double Statistics::DoCalculations()
{        
	dim3 dimGrid, dimBlock;

	StopWatchInterface *hTimer = NULL;
	sdkCreateTimer(&hTimer);
	hiprandState *devStates;

	checkCudaErrors(hipSetDevice(0));

    // Allocate memory on GPU    
	
	sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);
	
	hipMalloc((void **)&d_Means,  NBOOTSTRAPS * sizeof(float));
    hipMalloc((void **)&d_Data,  NSAMPLES * sizeof(float));
	hipMalloc((void **)&devStates, NBOOTSTRAPS * sizeof(hiprandState));
	
	// Copy data to GPU
	hipMemcpy(d_Data, h_Data, NSAMPLES * sizeof(float), hipMemcpyHostToDevice);

	// Copy meanprop to constant memory
	//hipMemcpyToSymbol(HIP_SYMBOL(c_MeanProp), h_MeanProp, NPar * sizeof(float), 0, hipMemcpyHostToDevice);
	
    // 512 threads per block
	threadsInX = 512;

    // Round up to get sufficient number of blocks
    blocksInX = (int)ceil((float)NBOOTSTRAPS / (float)threadsInX);
    
    dimGrid  = dim3(blocksInX, 1, 1);
    dimBlock = dim3(threadsInX, 1, 1);

	
	// Initialize states for random number generation
	SetupRandKernel<<<dimGrid, dimBlock>>>(devStates,NBOOTSTRAPS);

    // Do calculations	
	DoCalculationsGPU<<<dimGrid, dimBlock>>>(d_Means, d_Data, devStates, NBOOTSTRAPS, NSAMPLES);
	
	// Copy result to host
	hipMemcpy(h_Means, d_Means, NBOOTSTRAPS * sizeof(float), hipMemcpyDeviceToHost);

	checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    double gpuTime = 0.001 * sdkGetTimerValue(&hTimer);
    	
    // Free allocated memory on GPU
	hipFree( d_Means );
	hipFree( d_Data );
    hipFree( devStates );

	sdkDeleteTimer(&hTimer);
	
    hipDeviceReset();
	return gpuTime;
}

