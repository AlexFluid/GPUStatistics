#include "hip/hip_runtime.h"

#ifndef STATISTICS_KERNEL_CU_
#define STATISTICS_KERNEL_CU_

#include "statistics.h"
#include "help_functions.cu"

#include <hiprand/hiprand_kernel.h>

__global__ void SetupRandKernel(hiprandState *states, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx >= N)
		return;

    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(1234, idx, 0, &states[idx]);
}

__global__ void DoCalculationsGPU(float* Means, const float* __restrict__ Data, const hiprandState *states, int NBOOTSTRAPS, int NSAMPLES)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx >= NBOOTSTRAPS)
		return;

    hiprandState localState = states[idx];

	float Nf = (float)NSAMPLES - 1.0f;	
	float sum = 0.0f;
	for (int i = 0; i < NSAMPLES; i++)
	{
	    int randomIndex = (int)(hiprand_uniform(&localState) * Nf);
        sum += Data[randomIndex];
	}
	Means[idx] = sum / (float)NSAMPLES;
}

#endif