#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "statistics.h"
#include "statistics_kernel.cu"

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

Statistics::Statistics(int samples, int bootstraps)
{
    NSAMPLES = samples;
	NBOOTSTRAPS = bootstraps;
}

Statistics::Statistics(int samples, int bootstraps, int regressors)
{
    NSAMPLES = samples;
	NBOOTSTRAPS = bootstraps;
	NUMBER_OF_REGRESSORS = regressors;
}

Statistics::~Statistics()
{

}

void Statistics::SetInputDataPointers(float* data)
{
	h_Data = data;
}

void Statistics::SetInputDataPointers(float* fit, float* residuals, float* xtxxt)
{
	h_Fit = fit;
	h_Residuals = residuals;
	h_xtxxt = xtxxt;
}

void Statistics::SetOutputDataPointers(float* means)
{
	h_Means = means;
}

void Statistics::SetOutputDataPointersContrasts(float* contrasts)
{
	h_Contrasts = contrasts;
}


double Statistics::BootstrapMean()
{        
	dim3 dimGrid, dimBlock;

	StopWatchInterface *hTimer = NULL;
	sdkCreateTimer(&hTimer);
	
	checkCudaErrors(hipSetDevice(0));

    // Allocate memory on GPU    
	
	sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);
	
	hipMalloc((void **)&d_Means,  NBOOTSTRAPS * sizeof(float));
    hipMalloc((void **)&d_Data,  NSAMPLES * sizeof(float));
	
	// Copy data to GPU
	hipMemcpy(d_Data, h_Data, NSAMPLES * sizeof(float), hipMemcpyHostToDevice);
	
    // 256 threads per block
	threadsInX = 256;

    // Round up to get sufficient number of blocks
    blocksInX = (int)ceil((float)NBOOTSTRAPS / (float)threadsInX);
    
    dimGrid  = dim3(blocksInX, 1, 1);
    dimBlock = dim3(threadsInX, 1, 1);
	
    // Do calculations	
	BootstrapMeanGPU<<<dimGrid, dimBlock>>>(d_Means, d_Data, NBOOTSTRAPS, NSAMPLES);
	
	// Copy result to host
	hipMemcpy(h_Means, d_Means, NBOOTSTRAPS * sizeof(float), hipMemcpyDeviceToHost);

	checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    double gpuTime = 0.001 * sdkGetTimerValue(&hTimer);
    	
    // Free allocated memory on GPU
	hipFree( d_Means );
	hipFree( d_Data );
    
	sdkDeleteTimer(&hTimer);
	
    hipDeviceReset();
	return gpuTime;
}

double Statistics::BootstrapMeanCublas()
{        
	hipblasStatus_t status;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
		
	dim3 dimGrid, dimBlock;

	StopWatchInterface *hTimer = NULL;
	sdkCreateTimer(&hTimer);
	
	//checkCudaErrors(hipSetDevice(0));

    // Allocate memory on GPU    
	
	sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);
	
	hipMalloc((void **)&d_Means,  NBOOTSTRAPS * sizeof(float));
    hipMalloc((void **)&d_Data,   NSAMPLES * sizeof(float));
	hipMalloc((void **)&d_RandomSamples, NBOOTSTRAPS * NSAMPLES * sizeof(float));
	hipMalloc((void **)&d_MeanVector, NSAMPLES * sizeof(float));

	hipMemset(d_Means, 0, NBOOTSTRAPS * sizeof(float));

	// Copy data to GPU
	hipMemcpy(d_Data, h_Data, NSAMPLES * sizeof(float), hipMemcpyHostToDevice);
	
	
	// 256 threads per block
	threadsInX = 256;
	
    // Round up to get sufficient number of blocks
    blocksInX = (int)ceil((float)NSAMPLES / (float)threadsInX);
	
    dimGrid  = dim3(blocksInX, 1, 1);
    dimBlock = dim3(threadsInX, 1, 1);
	
	// Set mean vector
	SetMeanVector<<<dimGrid, dimBlock>>>(d_MeanVector, NSAMPLES);

	
	// 1024 threads per block
	threadsInX = 256;
	
    // Round up to get sufficient number of blocks
    blocksInX = (int)ceil((float)NBOOTSTRAPS / (float)threadsInX);
	
    dimGrid  = dim3(blocksInX, 1, 1);
    dimBlock = dim3(threadsInX, 1, 1);
				
    // Generate random samples
	GenerateRandomSamples<<<dimGrid, dimBlock>>>(d_RandomSamples, d_Data, NBOOTSTRAPS, NSAMPLES);
					
	// Perform matrix-vector multiplication, to calculate means
	float alpha = 1.0f;
	float beta = 0.0f;
	int stride = 1;
	int leadingDimension = NBOOTSTRAPS;
	status = hipblasSgemv(handle, HIPBLAS_OP_N, NBOOTSTRAPS, NSAMPLES, &alpha, d_RandomSamples, leadingDimension, d_MeanVector, stride, &beta, d_Means, stride);
	
	

	
	// Copy result to host
	hipMemcpy(h_Means, d_Means, NBOOTSTRAPS * sizeof(float), hipMemcpyDeviceToHost);

	
	

	checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    double gpuTime = 0.001 * sdkGetTimerValue(&hTimer);
    
	
		
    // Free allocated memory on GPU
	hipFree( d_Means );
	hipFree( d_MeanVector );
	hipFree( d_Data );
	hipFree( d_RandomSamples );
    
	sdkDeleteTimer(&hTimer);
	
	
	status = hipblasDestroy(handle);

    //hipDeviceReset();
		
	return gpuTime;
}
